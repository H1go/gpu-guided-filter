#include "hip/hip_runtime.h"
#include "box_filter.h"
#include<stdio.h>

__device__ void box_filter(float *in, float *out, int width, int height)
{
    int x = blockIdx.x * TILE_W + threadIdx.x - RADIUS;
    int y = blockIdx.y * TILE_H + threadIdx.y - RADIUS;
    const int idx = y * width + x;

    __shared__ float shMem[BLOCK_W][BLOCK_H];
    if(x<0 || y<0 || x>=width || y>=height) {
        shMem[threadIdx.x][threadIdx.y] = 0;
        return;
    }
    shMem[threadIdx.x][threadIdx.y] = in[idx];
   // printf("p_value: %f, im_value: %f, idx: %d\n", shMem[bindex], in[idx], idx);

    __syncthreads();

    if ((threadIdx.x >= RADIUS) && (threadIdx.x < (BLOCK_W - RADIUS)) &&
        (threadIdx.y >= RADIUS) && (threadIdx.y < (BLOCK_H - RADIUS))) {
        float sum = 0;
        for(int dy = -RADIUS; dy <= RADIUS; dy++) {
            for(int dx = -RADIUS; dx <= RADIUS; dx++) {
                float i = shMem[threadIdx.x + dx][threadIdx.y + dy];
                sum += i;
            }
        }
        out[idx] = sum / SIZE;
    }
}
