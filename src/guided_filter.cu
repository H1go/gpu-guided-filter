#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

#include "box_filter.h"
#include "math_kernels.h"

__device__ void compute_cov_var(float4 *mean_Ip, float4 *mean_II, float4 *mean_I,
        float4 *mean_p, float4 *var_I, float4 *cov_Ip,
        int width, int height)
{
    int x = blockIdx.x * TILE_W + threadIdx.x - RADIUS;
    int y = blockIdx.y * TILE_H + threadIdx.y - RADIUS;
    if (x >= 0 && y >= 0 && x < width && y < height) {
        int idx = y * width + x; 
        float4 m_I = mean_I[idx];
        var_I[idx] = mean_II[idx] - m_I * m_I;
        cov_Ip[idx] = mean_Ip[idx] - m_I * mean_p[idx];
    }
}

__device__ void compute_ab(float4 *var_I, float4 *cov_Ip, float4 *mean_I,
        float4 *mean_p, float4 *a, float4 *b, float eps,
        int width, int height)
{
    int x = blockIdx.x * TILE_W + threadIdx.x - RADIUS;
    int y = blockIdx.y * TILE_H + threadIdx.y - RADIUS;
    if (x >= 0 && y >= 0 && x < width && y < height) {
        int idx = y * width + x; 
        float4 a_ = cov_Ip[idx] / (var_I[idx] + eps);
        a[idx] = a_;
        b[idx] = mean_p[idx] - a_ * mean_I[idx];
    }
}

__device__ void compute_q(float4 *in, float4 *mean_a, float4 *mean_b, float4 *q,
        int width, int height)
{
    int x = blockIdx.x * TILE_W + threadIdx.x - RADIUS;
    int y = blockIdx.y * TILE_H + threadIdx.y - RADIUS;
    if (x >= 0 && y >= 0 && x < width && y < height) {
        int idx = y * width + x; 
        float4 im_ = in[idx];
        q[idx] = mean_a[idx] * im_ + mean_b[idx];
    }
}

__global__ void guidedFilterCudaKernel(float4* d_input,
        float4 *d_p,
        float4 *d_q,
        float4 *mean_I,
        float4 *mean_p,
        float4 *mean_Ip,
        float4 *mean_II,
        float4 *var_I,
        float4 *cov_Ip,
        float4 *a, 
        float4 *b,
        float4 *mean_a,
        float4 *mean_b,
        float4 *tmp,
        float4 *tmp2,
        int width, int height,
        float eps)
{
    mult(d_input, d_p, tmp, width, height);
    __syncthreads();
    mult(d_input, d_input, tmp2, width, height);

    __syncthreads();
    
    box_filter(d_input, mean_I, width, height);
    __syncthreads();
    box_filter(d_p, mean_p, width, height);
    __syncthreads();
    box_filter(tmp, mean_Ip, width, height);
    __syncthreads();
    box_filter(tmp2, mean_II, width, height);
    
    __syncthreads();

    compute_cov_var(mean_Ip, mean_II, mean_I, mean_p, var_I, cov_Ip, width, height);

    __syncthreads();
    
    compute_ab(var_I, cov_Ip, mean_I, mean_p, a, b, eps, width, height);
    
    __syncthreads();

    box_filter(a, mean_a, width, height);
    __syncthreads();
    box_filter(b, mean_b, width, height);

    __syncthreads();

    compute_q(d_p, mean_a, mean_b, d_q, width, height);
//    box_filter(d_input, d_q, width, height);
}

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void guidedFilterCuda(float4 *h_input,
        float4 *h_p,
        float4 *h_output,
        int width, int height,
        float eps)
{
       
    const int n = width * height * sizeof(float4);

    float4 *d_input, *d_p, *d_output, *mean_I, *mean_p,* mean_Ip,
          *mean_II, *var_I, *cov_Ip, *a, *b, *mean_a, *mean_b, *tmp, *tmp2;
    checkCudaErrors(hipMalloc<float4>(&d_input, n));
    checkCudaErrors(hipMalloc<float4>(&d_p, n));
    checkCudaErrors(hipMalloc<float4>(&d_output, n));
    checkCudaErrors(hipMalloc<float4>(&mean_I, n));
    checkCudaErrors(hipMalloc<float4>(&mean_p, n));
    checkCudaErrors(hipMalloc<float4>(&mean_Ip, n));
    checkCudaErrors(hipMalloc<float4>(&mean_II, n));
    checkCudaErrors(hipMalloc<float4>(&var_I, n));
    checkCudaErrors(hipMalloc<float4>(&cov_Ip, n));
    checkCudaErrors(hipMalloc<float4>(&a, n));
    checkCudaErrors(hipMalloc<float4>(&b, n));
    checkCudaErrors(hipMalloc<float4>(&mean_a, n));
    checkCudaErrors(hipMalloc<float4>(&mean_b, n));
    checkCudaErrors(hipMalloc<float4>(&tmp, n));
    checkCudaErrors(hipMalloc<float4>(&tmp2, n));

    checkCudaErrors(hipMemcpy(d_input, h_input, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_p, h_p, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_output, h_output, n, hipMemcpyHostToDevice));

    int GRID_W = width / TILE_W + 1;
    int GRID_H = height / TILE_H + 1;

    const dim3 block(BLOCK_W, BLOCK_H);
    const dim3 grid(GRID_W, GRID_H);
    //const dim3 grid(width/(block.x)+ block.x,height/(block.y)+block.y);
    //const dim3 grid((width + block.x-1)/block.x, (height + block.y - 1)/block.y);
    printf("grid_w: %d\n", grid.x);
    printf("grid_h: %d\n", grid.y);

    printf("block_w: %d\n", block.x);
    printf("block_h: %d\n", block.y);

    guidedFilterCudaKernel<<<grid, block>>>(d_input, d_p, d_output,
            mean_I, mean_p, mean_Ip, mean_II, var_I, cov_Ip, a, b,
            mean_a , mean_b, tmp, tmp2, width, height, eps);

    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(h_output, d_output, n, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_output));
    checkCudaErrors(hipFree(mean_I));
    checkCudaErrors(hipFree(mean_p));
    checkCudaErrors(hipFree(mean_Ip));
    checkCudaErrors(hipFree(mean_II));
    checkCudaErrors(hipFree(var_I));
    checkCudaErrors(hipFree(cov_Ip));
    checkCudaErrors(hipFree(a));
    checkCudaErrors(hipFree(b));
    checkCudaErrors(hipFree(mean_a));
    checkCudaErrors(hipFree(mean_b));
    checkCudaErrors(hipFree(tmp));
    checkCudaErrors(hipFree(tmp2));
}

void processUsingCuda(std::string input_file, std::string output_file) {
    cv::Mat input = cv::imread(input_file);
    if(input.empty()) {
        std::cout<<"Image Not Found: "<< input_file << std::endl;
        return;
    }

    cv::Mat inputRGBA;
    cvtColor(input, inputRGBA, CV_BGR2RGBA, 4);
    inputRGBA.convertTo(inputRGBA, CV_32FC4);
    inputRGBA /= 255;

    cv::Mat p = inputRGBA.clone();

    cv::Mat output (input.size(), inputRGBA.type());

    float eps = 0.2 * 0.2;

    //std::cout << inputGRAY << std::endl;
    guidedFilterCuda((float4*)inputRGBA.ptr<float4>(),
            (float4*)p.ptr<float4>(),
            (float4*)output.ptr<float4>(),
            inputRGBA.cols, inputRGBA.rows,
            eps);

   // std::cout << output << std::endl;
    output *= 255;

    //output.convertTo(output, CV_32F);
    cvtColor(output, output, CV_RGBA2BGR, 3);

    imwrite(output_file, output);
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Choose image" << std::endl;
        return 1;
    }
    processUsingCuda(argv[argc - 1], "out.png");
    return 0;
}
