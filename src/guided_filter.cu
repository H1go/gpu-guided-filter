#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

#include "box_filter.h"
#include "math_kernels.h"

__device__ void compute_cov_var(float *mean_Ip, float *mean_II, float *mean_I,
        float *mean_p, float *var_I, float *cov_Ip,
        int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x; 
    float m_I = mean_I[idx];
    var_I[idx] = max(mean_II[idx] - m_I * m_I, 0.);
    cov_Ip[idx] = max(mean_Ip[idx] - m_I * mean_p[idx], 0.);
}

__device__ void compute_ab(float *var_I, float *cov_Ip, float *mean_I,
        float *mean_p, float *a, float *b, float eps,
        int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x; 
    float a_ = cov_Ip[idx] / (var_I[idx] + eps);
    a[idx] = a_;
    b[idx] = mean_p[idx] - a_ * mean_I[idx];
}

__device__ void compute_q(float *in, float *mean_a, float *mean_b, float *q,
        int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x; 
    float im_ = in[idx];
    q[idx] = mean_a[idx] * im_ + mean_b[idx];
}

__global__ void mean_kernel(float* d_input,
        float *d_p,
        float *mean_I,
        float *mean_p,
        float *mean_Ip,
        float *mean_II,
        float *d_tmp,
        float *d_tmp2,
        int width, int height,
        float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        box_filter(d_input, mean_I, width, height);
        box_filter(d_p, mean_p, width, height);
        box_filter(d_tmp, mean_Ip, width, height);
        box_filter(d_tmp2, mean_II, width, height);
    }
}

__global__ void cov_var_ab_kernel(float* d_input,
        float *mean_I,
        float *mean_p,
        float *mean_Ip,
        float *mean_II,
        float *var_I,
        float *cov_Ip,
        float *a, 
        float *b,
        float *mean_a,
        float *mean_b,
        int width, int height,
        float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        compute_cov_var(mean_Ip, mean_II, mean_I, mean_p, var_I, cov_Ip, width, height);
        compute_ab(var_I, cov_Ip, mean_I, mean_p, a, b, eps, width, height);
    }
}

__global__ void output_kernel(float* d_input,
        float *d_p,
        float *d_q,
        float *a, 
        float *b,
        float *mean_a,
        float *mean_b,
        int width, int height,
        float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {

        box_filter(a, mean_a, width, height);
        box_filter(b, mean_b, width, height);
        compute_q(d_p, mean_a, mean_b, d_q, width, height);
        //d_q[y * width + x] = mean_a[y * width + x];
    }
}

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void guidedFilterCuda(float *h_input,
        float *h_p,
        float *h_output,
        float *h_tmp,
        float *h_tmp2,
        int width, int height,
        float eps)
{

    const int n = width * height * sizeof(float);

    float *d_input, *d_p, *d_output, *mean_I, *mean_p,* mean_Ip,
          *mean_II, *var_I, *cov_Ip, *a, *b, *mean_a, *mean_b, *d_tmp, *d_tmp2;
    checkCudaErrors(hipMalloc<float>(&d_input, n));
    checkCudaErrors(hipMalloc<float>(&d_p, n));
    checkCudaErrors(hipMalloc<float>(&d_output, n));
    checkCudaErrors(hipMalloc<float>(&mean_I, n));
    checkCudaErrors(hipMalloc<float>(&mean_p, n));
    checkCudaErrors(hipMalloc<float>(&mean_Ip, n));
    checkCudaErrors(hipMalloc<float>(&mean_II, n));
    checkCudaErrors(hipMalloc<float>(&var_I, n));
    checkCudaErrors(hipMalloc<float>(&cov_Ip, n));
    checkCudaErrors(hipMalloc<float>(&a, n));
    checkCudaErrors(hipMalloc<float>(&b, n));
    checkCudaErrors(hipMalloc<float>(&mean_a, n));
    checkCudaErrors(hipMalloc<float>(&mean_b, n));
    checkCudaErrors(hipMalloc<float>(&d_tmp, n));
    checkCudaErrors(hipMalloc<float>(&d_tmp2, n));

    checkCudaErrors(hipMemcpy(d_input, h_input, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_p, h_p, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_output, h_output, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_tmp, h_tmp, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_tmp2, h_tmp2, n, hipMemcpyHostToDevice));

    int GRID_W = ceil(width / (float)TILE_W);
    int GRID_H = ceil(height / (float)TILE_H);

    const dim3 block(TILE_W, TILE_H);
    const dim3 grid(GRID_W, GRID_H);
    //const dim3 grid(width/(block.x)+ block.x,height/(block.y)+block.y);
    //const dim3 grid((width + block.x-1)/block.x, (height + block.y - 1)/block.y);
    printf("grid_w: %d\n", grid.x);
    printf("grid_h: %d\n", grid.y);

    printf("block_w: %d\n", block.x);
    printf("block_h: %d\n", block.y);

    mean_kernel<<<grid, block>>>(d_input, d_p, mean_I, mean_p, mean_Ip,
            mean_II, d_tmp, d_tmp2, width, height, eps);

    hipDeviceSynchronize();

    cov_var_ab_kernel<<<grid, block>>>(d_input, mean_I, mean_p, mean_Ip,
        mean_II, var_I, cov_Ip, a, b, mean_a, mean_b, width, height,
        eps);

    hipDeviceSynchronize();

    output_kernel<<<grid, block>>>(d_input, d_p, d_output, a, b, mean_a, mean_b,
        width, height, eps);

    hipDeviceSynchronize();

    auto error = hipGetLastError();
    if (error != hipSuccess)
        printf("An error occured with CUDA: %s\n", hipGetErrorString(error));

    checkCudaErrors(hipMemcpy(h_output, d_output, n, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_output));
    checkCudaErrors(hipFree(mean_I));
    checkCudaErrors(hipFree(mean_p));
    checkCudaErrors(hipFree(mean_Ip));
    checkCudaErrors(hipFree(mean_II));
    checkCudaErrors(hipFree(var_I));
    checkCudaErrors(hipFree(cov_Ip));
    checkCudaErrors(hipFree(a));
    checkCudaErrors(hipFree(b));
    checkCudaErrors(hipFree(mean_a));
    checkCudaErrors(hipFree(mean_b));
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipFree(d_tmp2));
}

void processUsingCuda(std::string input_file, std::string output_file) {
    cv::Mat input = cv::imread(input_file);
    if(input.empty()) {
        std::cout<<"Image Not Found: "<< input_file << std::endl;
        return;
    }

    cv::Mat inputGRAY;
    cvtColor(input, inputGRAY, CV_BGR2GRAY, 1);
    inputGRAY.convertTo(inputGRAY, CV_32F);
    inputGRAY /= 255.f;

    cv::Mat p = inputGRAY.clone();

    cv::Mat output (input.size(), inputGRAY.type());

    float eps = 0.2 * 0.2;
    cv::Mat tmp = inputGRAY.mul(p);
    cv::Mat tmp2 = inputGRAY.mul(inputGRAY);

    //std::cout << inputGRAY << std::endl;
    guidedFilterCuda(inputGRAY.ptr<float>(),
            p.ptr<float>(),
            output.ptr<float>(),
            tmp.ptr<float>(),
            tmp2.ptr<float>(),
            inputGRAY.cols, inputGRAY.rows,
            eps);

    output *= 255;

    //std::cout << output << std::endl;
    //output.convertTo(output, CV_32F);
    cvtColor(output, output, CV_GRAY2BGR, 3);

    imwrite(output_file, output);
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Choose image" << std::endl;
        return 1;
    }
    processUsingCuda(argv[argc - 1], "out.png");
    return 0;
}
