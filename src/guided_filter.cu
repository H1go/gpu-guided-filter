#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <stdio.h>

#include "box_filter.h"

__device__ void compute_cov_var(float *mean_Ip, float *mean_II, float *mean_I,
        float *mean_p, float *var_I, float *cov_Ip,
        int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x; 
        float m_I = mean_I[idx];
        var_I[idx] = mean_Ip[idx] - m_I * m_I;
        cov_Ip[idx] = mean_II[idx] - m_I * mean_p[idx];
    }
}

__device__ void compute_ab(float *var_I, float *cov_Ip, float *mean_I,
        float *mean_p, float *a, float *b, float eps,
        int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x; 
        float a_ = cov_Ip[idx] / (var_I[idx] + eps);
        a[idx] = a_;
        b[idx] = mean_p[idx] - a_ * mean_I[idx];
    }
}

__device__ void compute_q(float *p, float *mean_a, float *mean_b, float *q,
        int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x; 
        float p_ = p[idx];
        q[idx] = mean_a[idx] * p_ + mean_b[idx];
    }
}

__global__ void guidedFilterCudaKernel(float* d_input,
        float *d_p,
        float *d_q,
        float *mean_I,
        float *mean_p,
        float *mean_Ip,
        float *mean_II,
        float *var_I,
        float *cov_Ip,
        float *a, 
        float *b,
        float *mean_a,
        float *mean_b,
        int width, int height,
        float eps)
{
    box_filter(d_input, mean_I, width, height);
    box_filter(d_input, mean_p, width, height);
    box_filter(d_input, mean_Ip, width, height);
    box_filter(d_input, mean_II, width, height);

    __syncthreads();

    compute_cov_var(mean_Ip, mean_II, mean_I, mean_p, var_I, cov_Ip, width, height);
    __syncthreads();
    compute_ab(var_I, cov_Ip, mean_I, mean_p, a, b, eps, width, height);
    __syncthreads();

    box_filter(a, mean_a, width, height);
    box_filter(b, mean_b, width, height);

    __syncthreads();

    compute_q(d_p, mean_a, mean_b, d_q, width, height);
}

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void guidedFilterCuda(float *h_input,
        float *h_p,
        float *h_output,
        int width, int height,
        float eps)
{
    const int n = width * height * sizeof(float);
    float *d_input, *d_p, *d_output, *mean_I, *mean_p,* mean_Ip,
          *mean_II, *var_I, *cov_Ip, *a, *b, *mean_a, *mean_b;
    checkCudaErrors(hipMalloc<float>(&d_input, n));
    checkCudaErrors(hipMalloc<float>(&d_p, n));
    checkCudaErrors(hipMalloc<float>(&d_output, n));
    checkCudaErrors(hipMalloc<float>(&mean_I, n));
    checkCudaErrors(hipMalloc<float>(&mean_p, n));
    checkCudaErrors(hipMalloc<float>(&mean_Ip, n));
    checkCudaErrors(hipMalloc<float>(&mean_II, n));
    checkCudaErrors(hipMalloc<float>(&var_I, n));
    checkCudaErrors(hipMalloc<float>(&cov_Ip, n));
    checkCudaErrors(hipMalloc<float>(&a, n));
    checkCudaErrors(hipMalloc<float>(&b, n));
    checkCudaErrors(hipMalloc<float>(&mean_a, n));
    checkCudaErrors(hipMalloc<float>(&mean_b, n));

    checkCudaErrors(hipMemcpy(d_input, h_input, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_p, h_p, n, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_output, h_output, n, hipMemcpyHostToDevice));

    int GRID_W = width / TILE_W + 1;
    int GRID_H = height / TILE_H + 1;

    const dim3 block(BLOCK_W, BLOCK_H);
    const dim3 grid(GRID_W, GRID_H);
    printf("grid_w: %d\n", grid.x);
    printf("grid_h: %d\n", grid.y);

    printf("block_w: %d\n", block.x);
    printf("block_h: %d\n", block.y);

    guidedFilterCudaKernel<<<grid,block>>>(d_input, d_p, d_output,
            mean_I, mean_p, mean_Ip, mean_II, var_I, cov_Ip, a, b,
            mean_a , mean_b, width, height, eps);

    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(h_output, d_output, n, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_p));
    checkCudaErrors(hipFree(d_output));
    checkCudaErrors(hipFree(mean_I));
    checkCudaErrors(hipFree(mean_p));
    checkCudaErrors(hipFree(mean_Ip));
    checkCudaErrors(hipFree(mean_II));
    checkCudaErrors(hipFree(var_I));
    checkCudaErrors(hipFree(cov_Ip));
    checkCudaErrors(hipFree(a));
    checkCudaErrors(hipFree(b));
    checkCudaErrors(hipFree(mean_a));
    checkCudaErrors(hipFree(mean_b));
}

void processUsingCuda(std::string input_file, std::string output_file) {
    cv::Mat input = cv::imread(input_file);
    if(input.empty()) {
        std::cout<<"Image Not Found: "<< input_file << std::endl;
        return;
    }

    cv::Mat inputGRAY;
    cvtColor(input, inputGRAY, CV_BGR2GRAY, 1);
    inputGRAY.convertTo(inputGRAY, CV_64F);
    inputGRAY /= 255;

    cv::Mat p = inputGRAY.clone();

    cv::Mat output (input.size(), inputGRAY.type());

    float eps = 0.2 * 0.2;

    guidedFilterCuda((float*) inputGRAY.ptr<float>(),
            (float*) p.ptr<float>(),
            (float*) output.ptr<float>(),
            inputGRAY.cols, inputGRAY.rows,
            eps);

    output *= 255;
    //output.convertTo(output, CV_32F);
    //cvtColor(output, output, CV_GRAY2BGR, 3);

    imwrite(output_file, output);
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Choose image" << std::endl;
        return 1;
    }
    processUsingCuda(argv[argc - 1], "out.png");
    return 0;
}
