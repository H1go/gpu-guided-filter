#include "hip/hip_runtime.h"
#include "math_kernels.h"
#include "box_filter.h"

__device__ float4 operator*(float4 a, float4 b)
{
	return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}

__device__ float4 operator/(float4 a, float4 b)
{
	return make_float4(a.x / b.x, a.y / b.y, a.z / b.z, a.w / b.w);
}

__device__ float4 operator/(float4 a, int b)
{
    return make_float4(a.x / b, a.y / b, a.z / b,  a.w / b);
}

__device__ float4 operator-(float4 a, float4 b)
{
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w);
}

__device__ float4 operator+(float4 a, float4 b)
{
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__device__ float4 operator+(float4 a, float b)
{
	return make_float4(a.x + b, a.y + b, a.z + b, a.w + b);
}

__device__ void mult(float4 *a, float4 *b, float4 *tmp, int width, int height)
{
    int x = blockIdx.x * TILE_W + threadIdx.x - RADIUS;
    int y = blockIdx.y * TILE_H + threadIdx.y - RADIUS;

    if (x < width && y < height) {
        int idx = y * width + x; 
        tmp[idx] = a[idx] * b[idx];
    }
}
/*
__device__ void pown_ (float4 *in, float4 *out, int n,  int width, int height)
{
    int x = blockIdx.x * TILE_W + threadIdx.x - RADIUS;
    int y = blockIdx.y * TILE_H + threadIdx.y - RADIUS;

    if (x < width && y < height) {
        int idx = y * width + x; 
        out[idx] = pow(in[idx], 2);
    }
}*/
